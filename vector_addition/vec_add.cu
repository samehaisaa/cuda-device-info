#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

__global__ void VecAdd(float* A, float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

void initializeVector(float* vec, int size) {
    for (int i = 0; i < size; i++) {
        vec[i] = static_cast<float>(rand()) / RAND_MAX; 
    }
}

int main() {
    int N = 1024; 
    size_t size = N * sizeof(float); 

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    initializeVector(h_A, N);
    initializeVector(h_B, N);

    float* d_A; 
    float* d_B; 
    float* d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    bool success = true;
    for (int i = 0; i < N; i++) {
        float expected = h_A[i] + h_B[i];
        if (fabs(h_C[i] - expected) > 1e-8) {
            success = false;
            std::cerr << "Error at index " << i << ": expected " << expected << ", got " << h_C[i] << std::endl;
        }
    }

    if (success) {
        std::cout << "All results are correct!" << std::endl;
    } else {
        std::cerr << "Some results are incorrect!" << std::endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
