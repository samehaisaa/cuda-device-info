#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "matrix_ops.h"

#define BLOCK_SIZE 16

__global__ void MatrixMulKernel(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < K) {
        float sum = 0.0f;
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = sum;
    }
}

void matrix_multiply(float* A, float* B, float* C, int M, int N, int K) {
    float *d_A, *d_B, *d_C;
    size_t size_A = M * N * sizeof(float);
    size_t size_B = N * K * sizeof(float);
    size_t size_C = M * K * sizeof(float);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((K + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                   (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    MatrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
    
    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}