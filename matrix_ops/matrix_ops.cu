#include "hip/hip_runtime.h"
#include <iostream>
#include "matrix_ops.h"

__global__ void MatrixMulKernel(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < K) {
        float sum = 0.0f;
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = sum;
    }
}

void matrix_multiply(float* A, float* B, float* C, int M, int N, int K) {
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * N * sizeof(float));
    hipMalloc(&d_B, N * K * sizeof(float));
    hipMalloc(&d_C, M * K * sizeof(float));

    hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * K * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    MatrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);

    hipMemcpy(C, d_C, M * K * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void print_matrix(float* matrix, int rows, int cols, const char* name) {
    std::cout << "Matrix " << name << " (" << rows << "x" << cols << "):" << std::endl;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            std::cout << matrix[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}